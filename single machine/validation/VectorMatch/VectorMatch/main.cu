
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "kernel.cu"
#include <glm/gtc/epsilon.hpp>
#include <random>

bool readFile(char *fileName, glm::vec3 **vecs, int *length);
int main(int argc, char** argv)
{
	if (argc==2)
	{
		int len = 1000;
		std::ofstream oFile;
		oFile.open(argv[1]);
		if (oFile.is_open())
		{
			oFile << len << "\n"; 
			std::default_random_engine rng;
			std::uniform_real_distribution<float> distribution(-100.0f, 100.0f);
			for (int i = 0; i < len;i++)
			{
				oFile << distribution(rng) << "," << distribution(rng) << "," << distribution(rng) << "\n";
			}
			oFile.close();
		}
		else
		{
			printf("Failed to open file: %s\n",argv[1]);
		}
		return 0;
	}
	if (argc < 3)
	{
		printf("Min 2 args, the two files to be compared, optional 3rd arg device id.");
		return 1;
	}

	int length1 = 0, length2 = 0;
	glm::vec3 *vecs1 = 0, *vecs2 = 0;
	std::ifstream file1;
	if (readFile(argv[1], &vecs1, &length1) && readFile(argv[2], &vecs2, &length2))
	{//Files are loaded
		if (length1 != length2)
		{
			printf("File lengths do not match: %d vs %d\n", length1, length2);
			return 1;
		}
        {
            //Sort both using GPU
            if (argc>=4)
            {
            	printf("Using CUDA device %d\n", atoi(argv[3]));
            	hipSetDevice(atoi(argv[3]));
            }
            glm::vec3 *d_vecs = nullptr;
            hipMalloc(&d_vecs, length1*sizeof(glm::vec3));
            hipMemcpy(d_vecs, vecs1, length1*sizeof(glm::vec3), hipMemcpyHostToDevice);
            thrust::sort(thrust::cuda::par, d_vecs, d_vecs + length1, vecCompare());
            hipMemcpy(vecs1, d_vecs, length1*sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipMemcpy(d_vecs, vecs2, length1*sizeof(glm::vec3), hipMemcpyHostToDevice);
            thrust::sort(thrust::cuda::par, d_vecs, d_vecs + length1, vecCompare());
            hipMemcpy(vecs2, d_vecs, length1*sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipFree(d_vecs); 
		}
        //Allocate storage for mismatches
        glm::vec3 *mismatchA = (glm::vec3 *)malloc(length1*sizeof(glm::vec3));
        glm::vec3 *mismatchB = (glm::vec3 *)malloc(length1*sizeof(glm::vec3));
        unsigned int mismatchCt = 0;
		//Compare both on CPU
		int j = 0;
		float minLength = FLT_MAX;
		float maxLength = 0;
		float meanLength = 0;
		int maxId = -1;
		for (int i = 0; i < length1;i++)
		{
			//printf("(%.9g,%.9g,%.9g)==(%.9g,%.9g,%.9g)\n", vecs1[i].x, vecs1[i].y, vecs1[i].z, vecs2[i].x, vecs2[i].y, vecs2[i].z);

			auto ret = glm::epsilonEqual(vecs1[i], vecs2[i], 1.0f);
			if (ret.x&&ret.y&&ret.z)
			{

				j++;
			}
            else
            {
                //Reorder components
                mismatchA[mismatchCt] = glm::vec3(vecs1[i].y, vecs1[i].x, vecs1[i].z);
                mismatchB[mismatchCt] = glm::vec3(vecs2[i].y, vecs2[i].x, vecs2[i].z);
                mismatchCt++;
            }
			float len = glm::length(vecs1[i] - vecs2[i]);
			minLength = len < minLength ? len : minLength;
			maxId = maxLength < len ? i : maxId;
			maxLength = maxLength < len ? len : maxLength;
			meanLength += len / length1;
		}
		//printf("%d/%d Successful matches!\n", j, length1);
		//printf("Min diff: %f\nMax diff: %f (%d)\nMean diff:%f\n", minLength, maxLength, maxId, meanLength);
		//printf("(%.9g,%.9g,%.9g)==(%.9g,%.9g,%.9g)\n", vecs1[maxId].x, vecs1[maxId].y, vecs1[maxId].z, vecs2[maxId].x, vecs2[maxId].y, vecs2[maxId].z);

        if (j!=length1)
        {//If first sort failed, resort the mismatched vectors
            glm::vec3 *d_vecs = nullptr;
            hipMalloc(&d_vecs, mismatchCt*sizeof(glm::vec3));
            hipMemcpy(d_vecs, mismatchA, mismatchCt*sizeof(glm::vec3), hipMemcpyHostToDevice);
            thrust::sort(thrust::cuda::par, d_vecs, d_vecs + mismatchCt, vecCompare());
            hipMemcpy(mismatchA, d_vecs, mismatchCt*sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipMemcpy(d_vecs, mismatchB, mismatchCt*sizeof(glm::vec3), hipMemcpyHostToDevice);
            thrust::sort(thrust::cuda::par, d_vecs, d_vecs + mismatchCt, vecCompare());
            hipMemcpy(mismatchB, d_vecs, mismatchCt*sizeof(glm::vec3), hipMemcpyDeviceToHost);
            hipFree(d_vecs);
            //Re match remainders
            minLength = FLT_MAX;
            maxLength = 0;
            meanLength = 0;
            maxId = -1;
            for (int i = 0; i < mismatchCt; i++)
            {
                auto ret = glm::epsilonEqual(mismatchA[i], mismatchB[i], 1.0f);
                if (ret.x&&ret.y&&ret.z)
                {
                    j++;
                }
                float len = glm::length(mismatchA[i] - mismatchB[i]);
                minLength = len < minLength ? len : minLength;
                maxId = maxLength < len ? i : maxId;
                maxLength = maxLength < len ? len : maxLength;
                meanLength += len / length1;
            }
            //printf("----------\n");
            printf("%d/%d Successful matches!\n", j, length1);
            printf("Min diff: %f\nMax diff: %f\nMean diff:%f\n", minLength, maxLength, meanLength);
            printf("(%.9g,%.9g,%.9g)==(%.9g,%.9g,%.9g)\n", vecs1[maxId].x, vecs1[maxId].y, vecs1[maxId].z, vecs2[maxId].x, vecs2[maxId].y, vecs2[maxId].z);
        }
	}
	else
	{
		return 1;
	}
	free(vecs1);
	free(vecs2);
	return 0;
}

bool readFile(char *fileName, glm::vec3 **vecs, int *length)
{
	char line[1024];
	std::ifstream file;
	file.open(fileName);
	if (file.is_open())
	{
		file.getline(&line[0], 1024);
		sscanf(line, "%d", length);
		if (*length <= 0)
		{
			printf("File does not begin with a positive integer\n");
			return false;
		}
		*vecs = (glm::vec3*)malloc(*length*sizeof(glm::vec3));
		for (int i = 0; i < *length; i++)
		{
			file.getline(&line[0], 1024);
			if (sscanf(line, "%f,%f,%f", &(*vecs)[i].x, &(*vecs)[i].y, &(*vecs)[i].z)<3)
			{
				printf("Line %d of File was not recognised.\n", i);
				return false;
			}
		}
		file.close();
	}
	else
	{
		printf("Could not open file: %s\n", fileName);
		return false;
	}
	return true;
}