
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "glm/glm.hpp"
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>

struct vecCompare : public thrust::binary_function < const glm::vec3, const glm::vec3, bool >
{
	__host__ __device__ bool operator()(const glm::vec3 &lhs, const glm::vec3 &rhs) const {
		if (lhs.x == rhs.x)
		{
			if (lhs.y == rhs.y)
			{
				return (lhs.z < rhs.z);
			}
			else
			{
				return (lhs.y < rhs.y);
			}
		}
		else
		{
			return (lhs.x < rhs.x);
		}
	}
};